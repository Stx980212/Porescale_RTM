#include "hip/hip_runtime.h"
#include "reactions2d.cuh"
#include "fvm_utils.cuh"
#include "cuda_utils.cuh"  // Include the new header
#include <iostream>

__device__ void computeReactionRates(
    double* rates,
    const double* concentrations,
    ReactionType reaction_type,
    const ReactionParameters& params
) {
    double cA = concentrations[0];  // Species A concentration
    double cB = concentrations[1];  // Species B concentration
    double cC = concentrations[2];  // Species C concentration

    double forward_rate = 0.0f;
    double backward_rate = 0.0f;
    double deviation = 0.0f;
    double relaxation_rate = 0.0f;
    
    switch(reaction_type) {
        case ReactionType::A_PLUS_B_TO_C:
            // Simple second-order reaction: A + B -> C
            forward_rate = params.k_forward * cA * cB;
            backward_rate = 0.0f;
            rates[0] = -forward_rate + backward_rate;                  // Rate for A
            rates[1] = -forward_rate + backward_rate;                  // Rate for B
            rates[2] = forward_rate - backward_rate;                   // Rate for C
            break;
            
        case ReactionType::NONLINEAR:
            // Nonlinear reaction with backward reaction: A + B <-> C
            forward_rate = params.k_forward * cA * cB;
            backward_rate = params.k_backward * cC;
            rates[0] = -forward_rate + backward_rate;                  // Rate for A
            rates[1] = -forward_rate + backward_rate;                  // Rate for B
            rates[2] = forward_rate - backward_rate;                   // Rate for C
            break;
            
        case ReactionType::EQUILIBRIUM:
            // Fast equilibrium approximation
            deviation = cA * cB - params.equilibrium_K * cC;
            relaxation_rate = params.k_forward * deviation;
            rates[0] = -relaxation_rate;                              // Rate for A
            rates[1] = -relaxation_rate;                              // Rate for B
            rates[2] = relaxation_rate;                               // Rate for C
            break;
    }
}

__global__ void computeReactions2D(
    double* concentrations,
    int nx,
    int ny,
    double dt,
    int num_species,
    ReactionType reaction_type,
    ReactionParameters params
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= nx || j >= ny) return;
    
    // Get index for current cell
    int idx = (j * nx + i) * num_species;
    
    // Local array for species concentrations
    double local_conc[3];
    double reaction_rates[3];
    
    // Load concentrations
    for (int s = 0; s < num_species; s++) {
        local_conc[s] = concentrations[idx + s];
    }
    
    // Compute reaction rates
    computeReactionRates(reaction_rates, local_conc, reaction_type, params);
    
    // Update concentrations using semi-implicit scheme
    for (int s = 0; s < num_species; s++) {
        double new_conc = local_conc[s] + dt * reaction_rates[s];
        
        // Ensure positivity
        new_conc = fmaxf(0.0f, new_conc);
        
        // Write back to global memory
        concentrations[idx + s] = new_conc;
    }
    
    // Add stability check for debugging
    #ifdef DEBUG
    for (int s = 0; s < num_species; s++) {
        if (isnan(concentrations[idx + s]) || isinf(concentrations[idx + s])) {
            printf("Warning: Invalid concentration at (%d,%d) species %d\n", i, j, s);
        }
    }
    #endif
}

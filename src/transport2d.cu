#include "hip/hip_runtime.h"
#include "transport2d.cuh"
#include "fvm_utils.cuh"
#include "cuda_utils.cuh" 
#include <iostream>
#include <cstring>

// Calculate fluxes at cell interfaces
__global__ void calculateFluxesKernel(
    const float* concentrations,    // Cell-averaged concentrations
    float* fluxes_x,               // Fluxes at x-interfaces
    float* fluxes_y,               // Fluxes at y-interfaces
    const int* mask,      
    const float* modified_diffusion,  // Add modified_diffusion parameter
    bool has_modified_diffusion,      // Add flag for modified diffusion

    int nx, int ny,
    float dx, float dy,
    float dt,
    int num_species,
    float2 velocity,
    float2 diffusion
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate x-direction fluxes
    if (i < nx+1 && j < ny) {  // +1 for interfaces
        for (int s = 0; s < num_species; s++) {
            // no-flux boundary
            if (i == 0 || i == nx || 
                (i > 0 && i < nx && (!mask[(j * nx + i-1)] || !mask[(j * nx + i)]))) {
                fluxes_x[(j * (nx+1) + i) * num_species + s] = 0.0f;
                continue;
            }

            float cL = concentrations[(j * nx + i-1) * num_species + s];
            float cR = concentrations[(j * nx + i) * num_species + s];
            
            // MUSCL reconstruction
            //float cL_interface, cR_interface;
            //FVMUtils::Reconstruction::muscl(cL, cR, dx, cL_interface, cR_interface);
            
            // Calculate advective flux using upwind scheme
            float flux_adv = FVMUtils::NumericalFlux::upwind(cL, cR, velocity.x);

            // Get appropriate diffusion coefficient
            float diff_coef;
            if (has_modified_diffusion) {
                // Average diffusion coefficients of adjacent cells
                float diff_L = modified_diffusion[j * nx + i-1];
                float diff_R = modified_diffusion[j * nx + i];
                diff_coef = 2.0f * (diff_L * diff_R) / (diff_L + diff_R + 1e-20f);
            } else {
                diff_coef = diffusion.x;
            }

            // Calculate diffusive flux with modified coefficient
            float flux_diff = FVMUtils::NumericalFlux::diffusive(cL, cR, dx, diff_coef);
            
            // Store total flux
            fluxes_x[(j * (nx+1) + i) * num_species + s] = flux_adv + flux_diff;
        }
    }
    
    // Calculate y-direction fluxes
    if (i < nx && j < ny+1) {
        for (int s = 0; s < num_species; s++) {
            // At domain boundaries, set flux to zero
            if (j == 0 || j == ny || 
                (j > 0 && j < ny && (!mask[((j-1) * nx + i)] || !mask[(j * nx + i)]))) {
                fluxes_y[(j * nx + i) * num_species + s] = 0.0f;
                continue;
            }
            
            float cB = concentrations[((j-1) * nx + i) * num_species + s];
            float cT = concentrations[(j * nx + i) * num_species + s];
            
            // Calculate advective and diffusive fluxes for interior points
            float flux_adv = FVMUtils::NumericalFlux::upwind(cB, cT, velocity.y);

            // Get appropriate diffusion coefficient
            float diff_coef;
            if (has_modified_diffusion) {
                // Average diffusion coefficients of adjacent cells
                float diff_B = modified_diffusion[(j-1) * nx + i];
                float diff_T = modified_diffusion[j * nx + i];
                diff_coef = 2.0f * (diff_B * diff_T) / (diff_B + diff_T + 1e-20f);
            } else {
                diff_coef = diffusion.y;
            }

            // Calculate diffusive flux with modified coefficient
            float flux_diff = FVMUtils::NumericalFlux::diffusive(cB, cT, dy, diff_coef);
            
            fluxes_y[(j * nx + i) * num_species + s] = flux_adv + flux_diff;
        }
    }
}

// Update cell averages using calculated fluxes
__global__ void updateConcentrationsKernel(
    float* concentrations_new,
    const float* concentrations,
    const float* fluxes_x,
    const float* fluxes_y,
    const float* cell_volumes,
    int nx, int ny,
    float dx, float dy,
    float dt,
    int num_species
) {
    const float UNDER_RELAX = 0.5f;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= nx || j >= ny) return;
    
    for (int s = 0; s < num_species; s++) {
        int idx = (j * nx + i) * num_species + s;
        
        // Get fluxes at all cell interfaces
        float flux_left = fluxes_x[(j * (nx+1) + i) * num_species + s];
        float flux_right = fluxes_x[(j * (nx+1) + i + 1) * num_species + s];
        float flux_bottom = fluxes_y[(j * nx + i) * num_species + s];
        float flux_top = fluxes_y[((j+1) * nx + i) * num_species + s];

        // Calculate mass change
        float dmass = -dt * (
            (flux_right - flux_left) * dy +  // Mass flux through x-faces
            (flux_top - flux_bottom) * dx    // Mass flux through y-faces
        );

        // Update concentration based on mass change and cell volume
        float new_mass = concentrations[idx] * cell_volumes[j * nx + i] + dmass;
        float new_conc = new_mass / cell_volumes[j * nx + i];

        // Apply underrelaxation
        concentrations_new[idx] = UNDER_RELAX * new_conc + 
                                 (1.0f - UNDER_RELAX) * concentrations[idx];

        // Ensure positivity
        concentrations_new[idx] = fmaxf(0.0f, concentrations_new[idx]);
    }
}


TransportSolver2D::TransportSolver2D(
    int nx, int ny, float dx, float dy, float dt, int num_species)
    : nx_(nx), ny_(ny), dx_(dx), dy_(dy), dt_(dt), num_species_(num_species), d_mask_(nullptr), d_modified_diffusion_(nullptr), 
      has_modified_diffusion_(false) {
    // Allocate device memory
    checkCudaErrors(hipMalloc(&d_concentrations_, nx * ny * num_species * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_concentrations_new_, nx * ny * num_species * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_fluxes_x_, (nx+1) * ny * num_species * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_fluxes_y_, nx * (ny+1) * num_species * sizeof(float)));
     
    // Allocate mask memory
    checkCudaErrors(hipMalloc(&d_mask_, nx * ny * sizeof(int)));
    // Initialize mask to all valid (1)
    checkCudaErrors(hipMemset(d_mask_, 1, nx * ny * sizeof(int)));

    // Allocate modified diffusion coefficients memory
    checkCudaErrors(hipMalloc(&d_modified_diffusion_, nx * ny * sizeof(float)));

    // Initialize with default diffusion values
    std::vector<float> default_diffusion(nx * ny, diffusion_.x);
    checkCudaErrors(hipMemcpy(d_modified_diffusion_, default_diffusion.data(),
                              nx * ny * sizeof(float), hipMemcpyHostToDevice));

    // Allocate memory for cell volumes
    checkCudaErrors(hipMalloc(&d_cell_volumes_, nx * ny * sizeof(float)));
    
    // Initialize with unit volumes
    std::vector<float> unit_volumes(nx * ny, 1.0f);
    checkCudaErrors(hipMemcpy(d_cell_volumes_, unit_volumes.data(),
                              nx * ny * sizeof(float), hipMemcpyHostToDevice));

    // Initialize parameters
    velocity_ = make_float2(0.0f, 0.0f);
    diffusion_ = make_float2(0.001f, 0.001f);
}

TransportSolver2D::~TransportSolver2D() {
    checkCudaErrors(hipFree(d_concentrations_));
    checkCudaErrors(hipFree(d_concentrations_new_));
    checkCudaErrors(hipFree(d_fluxes_x_));
    checkCudaErrors(hipFree(d_fluxes_y_));
    if (d_mask_) {
        checkCudaErrors(hipFree(d_mask_));
    }
    if (d_modified_diffusion_) {
        checkCudaErrors(hipFree(d_modified_diffusion_));
    }
    if (d_cell_volumes_) {
        checkCudaErrors(hipFree(d_cell_volumes_));
    }
}

std::vector<float> TransportSolver2D::getDiffusionCoefficients() const {
    if (!has_modified_diffusion_) {
        // Return uniform diffusion coefficients
        return std::vector<float>(nx_ * ny_, diffusion_.x);  // Assuming isotropic diffusion
    }

    // Return modified diffusion coefficients
    std::vector<float> host_diffusion(nx_ * ny_);
    checkCudaErrors(hipMemcpy(host_diffusion.data(), d_modified_diffusion_,
                              nx_ * ny_ * sizeof(float),
                              hipMemcpyDeviceToHost));
    return host_diffusion;
}

void TransportSolver2D::setModifiedDiffusion(const std::vector<float>& modified_diffusion) {
    if (modified_diffusion.size() != nx_ * ny_) {
        throw std::runtime_error("Modified diffusion array size does not match domain dimensions");
    }

    // Allocate device memory if not already done
    if (!has_modified_diffusion_) {
        checkCudaErrors(hipMalloc(&d_modified_diffusion_, nx_ * ny_ * sizeof(float)));
    }

    checkCudaErrors(hipMemcpy(d_modified_diffusion_, modified_diffusion.data(),
                              nx_ * ny_ * sizeof(float),
                              hipMemcpyHostToDevice));
    has_modified_diffusion_ = true;
}

void TransportSolver2D::setCellVolumes(const std::vector<float>& volumes) {
    if (volumes.size() != nx_ * ny_) {
        throw std::runtime_error("Cell volumes array size does not match domain dimensions");
    }
    
    checkCudaErrors(hipMemcpy(d_cell_volumes_, volumes.data(),
                              nx_ * ny_ * sizeof(float),
                              hipMemcpyHostToDevice));
}

std::vector<float> TransportSolver2D::getCellVolumes() const {
    std::vector<float> host_volumes(nx_ * ny_);
    checkCudaErrors(hipMemcpy(host_volumes.data(), d_cell_volumes_,
                              nx_ * ny_ * sizeof(float),
                              hipMemcpyDeviceToHost));
    return host_volumes;
}

float TransportSolver2D::getTotalMass() const {
    std::vector<float> host_concentrations(nx_ * ny_ * num_species_);
    std::vector<float> host_volumes(nx_ * ny_);
    
    checkCudaErrors(hipMemcpy(host_concentrations.data(), d_concentrations_,
                              nx_ * ny_ * num_species_ * sizeof(float),
                              hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(host_volumes.data(), d_cell_volumes_,
                              nx_ * ny_ * sizeof(float),
                              hipMemcpyDeviceToHost));
    
    float total_mass = 0.0f;
    for (int i = 0; i < nx_ * ny_; ++i) {
        for (int s = 0; s < num_species_; ++s) {
            total_mass += host_concentrations[i * num_species_ + s] * host_volumes[i];
        }
    }
    return total_mass;
}

void TransportSolver2D::solve(std::vector<float>& concentrations) {
    checkCFLCondition();
    // Copy data to device
    checkCudaErrors(hipMemcpy(d_concentrations_, concentrations.data(),
               nx_ * ny_ * num_species_ * sizeof(float),
               hipMemcpyHostToDevice));
    
    // Set up grid and block dimensions
    dim3 block_size(16, 16);
    dim3 num_blocks_fluxes(
        (nx_ + 2 + block_size.x - 1) / block_size.x,
        (ny_ + 1 + block_size.y - 1) / block_size.y
    );
    dim3 num_blocks_update(
        (nx_ + block_size.x - 1) / block_size.x,
        (ny_ + block_size.y - 1) / block_size.y
    );
    
    // Step 1: Calculate fluxes at cell interfaces
    calculateFluxesKernel<<<num_blocks_fluxes, block_size>>>(
        d_concentrations_,
        d_fluxes_x_,
        d_fluxes_y_,
        d_mask_,
        d_modified_diffusion_,  // Add modified diffusion array
        has_modified_diffusion_, // Add flag for using modified diffusion
        nx_, ny_,
        dx_, dy_,
        dt_,
        num_species_,
        velocity_,
        diffusion_
    );
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkBoundaryFluxes();
        
    // Step 2: Update cell averages using computed fluxes
    updateConcentrationsKernel<<<num_blocks_update, block_size>>>(
        d_concentrations_new_,
        d_concentrations_,
        d_fluxes_x_,
        d_fluxes_y_,
        d_cell_volumes_,
        nx_, ny_,
        dx_, dy_,
        dt_,
        num_species_
    );
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
        
    // Copy results back to host
    checkCudaErrors(hipMemcpy(concentrations.data(), d_concentrations_new_,
               nx_ * ny_ * num_species_ * sizeof(float),
               hipMemcpyDeviceToHost));
    
    // Swap pointers for next iteration
    float* temp = d_concentrations_;
    d_concentrations_ = d_concentrations_new_;
    d_concentrations_new_ = temp;
}

void TransportSolver2D::setVelocity(float vx, float vy) {
    velocity_ = make_float2(vx, vy);
}

void TransportSolver2D::setDiffusion(float dx, float dy) {
    diffusion_ = make_float2(dx, dy);
}

void TransportSolver2D::checkCFLCondition() {
    float max_velocity = fmax(abs(velocity_.x), abs(velocity_.y));
    float diff_cfl = (diffusion_.x/(dx_*dx_) + diffusion_.y/(dy_*dy_)) * dt_;
    float adv_cfl = max_velocity * dt_ / fmin(dx_, dy_);
    
    if (diff_cfl > 0.25 || adv_cfl > 1.0) {  // Changed from 0.5 to 0.25 for diffusion
        std::cerr << "Warning: CFL condition might be violated\n"
                  << "Diffusive CFL: " << diff_cfl << "\n"
                  << "Advective CFL: " << adv_cfl << std::endl;
    }
}

void TransportSolver2D::checkBoundaryFluxes() {
    std::vector<float> h_fluxes_x((nx_+1) * ny_ * num_species_);
    std::vector<float> h_fluxes_y(nx_ * (ny_+1) * num_species_);
    
    // Copy fluxes back to host for checking
    hipMemcpy(h_fluxes_x.data(), d_fluxes_x_, 
               h_fluxes_x.size() * sizeof(float), 
               hipMemcpyDeviceToHost);
    hipMemcpy(h_fluxes_y.data(), d_fluxes_y_, 
               h_fluxes_y.size() * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Check x-direction boundary fluxes
    for (int j = 0; j < ny_; j++) {
        for (int s = 0; s < num_species_; s++) {
            float left_flux = h_fluxes_x[(j * (nx_+1)) * num_species_ + s];
            float right_flux = h_fluxes_x[(j * (nx_+1) + nx_) * num_species_ + s];
            if (std::abs(left_flux) > 1e-10 || std::abs(right_flux) > 1e-10) {
                std::cout << "Warning: Non-zero boundary flux detected in x-direction\n";
                std::cout << "j=" << j << ", species=" << s 
                         << ", left=" << left_flux 
                         << ", right=" << right_flux << std::endl;
            }
        }
    }
    
    // Check y-direction boundary fluxes
    for (int i = 0; i < nx_; i++) {
        for (int s = 0; s < num_species_; s++) {
            float bottom_flux = h_fluxes_y[i * num_species_ + s];
            float top_flux = h_fluxes_y[(ny_ * nx_ + i) * num_species_ + s];
            if (std::abs(bottom_flux) > 1e-10 || std::abs(top_flux) > 1e-10) {
                std::cout << "Warning: Non-zero boundary flux detected in y-direction\n";
                std::cout << "i=" << i << ", species=" << s 
                         << ", bottom=" << bottom_flux 
                         << ", top=" << top_flux << std::endl;
            }
        }
    }
}

void TransportSolver2D::setMask(const std::vector<int>& mask) {
    if (mask.size() != nx_ * ny_) {
        throw std::runtime_error("Mask size does not match domain dimensions");
    }
    
    // Copy mask to device
    checkCudaErrors(hipMemcpy(d_mask_, mask.data(), 
                              nx_ * ny_ * sizeof(int), 
                              hipMemcpyHostToDevice));
}

std::vector<int> TransportSolver2D::getMask() const {
    std::vector<int> host_mask(nx_ * ny_);
    checkCudaErrors(hipMemcpy(host_mask.data(), d_mask_,
                             nx_ * ny_ * sizeof(int),
                             hipMemcpyDeviceToHost));
    return host_mask;
}